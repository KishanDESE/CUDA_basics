#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 30




void normal_cpu_fun(int *a){
	for(int i =0; i < N; i++)
		a[i] = i*i;

}


__global__ void gpufun(int *a){

	a[threadIdx.x] = threadIdx.x * threadIdx.x;

}


int main(){
	int a[N]; //Can not pass directly array in CPU memory to GPU memory

	int * da; //Pointer on CPU pointing memory on GPU

	hipMalloc(&da,N*sizeof(int));
	
	gpufun<<<1,N>>>(da); //da passed by value

	hipMemcpy(a,da,N*sizeof(int),hipMemcpyDeviceToHost); //Transfer data cpu to gpu or gpu to cpu

	for(int i=0;i<N;i++)
		printf("%d\t",a[i]); //done on cpu

//No cudaDeviceSynchronize
//Because cudaMemcpy will will block CPU proccess to get GPU data.
//cudaMemcpy is dependent on PCIe express bus


return 0;
}

