#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 30


__global__ void gpufun(int *a){

  a[threadIdx.x] = threadIdx.x * threadIdx.x;

}


int main(){
  int * da; 

	hipMalloc(&da,N*sizeof(int));//lets allocate memory in GPU


  gpufun<<<1,N>>>(da); 

	
  for(int i=0;i<N;i++)
      printf("%d\t", da[i]);

  return 0;
}

