#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(){ //__global__ indicate it is not normal kernel function but for GPU
printf("Hello world on GPU \n");
}


int main (){
	dkernel <<<1,1>>>(); //<<<no. of blocks,no. of threads in block>>>
	dkernel <<<1,1>>>(); //Kernels (by default) are executed one after another. CPU launches them and move ahead.
	

	dkernel<<<1,32>>>(); //1 block 32 threads





		printf("Hello world 1 on CPU \n"); //CPU is not waiting for syncing GPU buffer. So anywhere possible. But mostly CPU operate faster kernel so print is before.

	hipDeviceSynchronize(); //Tells GPU to do all work than synchronize GPU buffer with CPU.


		printf("Hello world 2 on CPU \n"); //After sync it will first executed GPU part than next lines of main code. 

return 0;

}
