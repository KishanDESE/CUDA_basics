#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 30

__global__ void gpufun(int *a) {
    // ThreadIdx.x gives the unique identifier of each thread within a block
    a[threadIdx.x] = threadIdx.x * threadIdx.x;
}

int main() {

//Unified Memory: Even though Jetson Nano supports unified memory, you need to use specific CUDA functions to allocate unified memory.

    int *a;

    // Allocate unified memory that can be accessed by both the host and the device
    hipMallocManaged(&a, N * sizeof(int));

    gpufun<<<1, N>>>(a); // Launch the kernel with one block and N threads

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Access and print the results directly from the unified memory
    for(int i = 0; i < N; i++)
        printf("%d\t", a[i]);

    // Free the unified memory
    hipFree(a);

    return 0;
}

