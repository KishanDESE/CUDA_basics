#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 30


__global__ void gpufun(int *a){

  // ThreadIdx.x gives the unique identifier of each thread within a block
  a[threadIdx.x] = threadIdx.x * threadIdx.x;

}


int main(){
  int a[N]; // Allocate memory on host (CPU) that can be accessed by GPU

  gpufun<<<1,N>>>(a); // Launch the kernel with one block and N threads

  // Access and print the results directly from the host memory (a)
  for(int i=0;i<N;i++)
      printf("%d\t", a[i]);

  return 0;
}

