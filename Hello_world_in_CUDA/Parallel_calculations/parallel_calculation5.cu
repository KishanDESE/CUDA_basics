/*

Limits on the Jetson Nano


The specific limits for the Jetson Nano are determined by its CUDA capability. The Jetson Nano has a Maxwell GPU with CUDA Compute Capability 5.3. Here are the typical limits:

Threads per Block: Up to 1024 threads per block.
Block Dimensions:
In each dimension, a block can have a maximum size of 1024 threads (x, y, z).
Typical configurations are up to (1024, 1, 1) for 1D, (32, 32, 1) for 2D, and (16, 16, 4) for 3D blocks.
Grid Dimensions:
The maximum size of a grid dimension can be up to 2^31-1 (2147483647) in each dimension, but practical limits are based on the available GPU resources and memory.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(){
if(	threadIdx.x==0&&blockIdx.x==0&&
	threadIdx.y==0&&blockIdx.y==0&&
	threadIdx.z==0&&blockIdx.z==0){
		printf("gridDim.x : %d,gridDim.y : %d,gridDim.z %d\nblockDim.x : %d,blockDim.y : %d,blockDim.z : %d\n",
gridDim.x,gridDim.y,gridDim.z,
blockDim.x,blockDim.y,blockDim.z);
}
}
int main(){
	dim3 grid(2,3,4);	//2*3*4 blocks
	dim3 block(1,5,3);	//1*5*3 threads

	dkernel<<<grid,block>>>();
	hipDeviceSynchronize();
	

return 0;
}
